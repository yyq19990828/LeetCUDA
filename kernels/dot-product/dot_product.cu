#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>
#include <torch/types.h>
#include <vector>

#define WARP_SIZE 32
/**
 * @brief 将给定变量的地址强制类型转换为 int4 指针，并获取其第一个 int4 元素。
 *
 * 该宏定义用于将任意类型的变量 value 的地址 reinterpret_cast（重新解释）为 int4* 类型，
 * 然后通过 [0] 取出第一个 int4 元素。int4 是 CUDA 中的一个结构体，包含四个 int 分量（x, y, z, w），
 * 常用于向量化操作以提升内存访问效率。
 *
 * 注意事项：
 * 1. value 变量的内存布局必须满足 int4 对齐要求（16 字节对齐），否则可能导致未定义行为。
 * 2. 该宏通常用于需要将数据块按 int4 方式批量处理的场景，如内存拷贝或向量化计算。
 * 3. 使用时需确保 value 的类型和大小适合 int4 的 reinterpret_cast 操作。
 */
//* reinterpret_cast<...>适用于指向一个完全不同、但内存布局兼容
//*（或你明确知道如何处理其布局）的类型的指针时, 可以通过一次操作
//* 处理多个数据
#define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])

// -------------------------------------- FP32
// -------------------------------------- Warp Reduce Sum
template <const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f32(float val) {
#pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

// Dot Product
// grid(N/256), block(256)
// a: Nx1, b: Nx1, y=sum(elementwise_mul(a,b))
template <const int NUM_THREADS = 256>
__global__ void dot_prod_f32_f32_kernel(float *a, float *b, float *y, int N) {
  int tid = threadIdx.x; //* 线程快内的相对位置
  int idx = blockIdx.x * NUM_THREADS + tid; //* 绝对位置
  
  //* constexpr 表示编译器在编译时就能计算出值
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  //* __shared__ 表示共享内存, 对于每个线程块, 共享内存是共享的
  __shared__ float reduce_smem[NUM_WARPS];

  // keep the data in register is enough for warp operaion.
  float prod = (idx < N) ? a[idx] * b[idx] : 0.0f;
  int warp = tid / WARP_SIZE; //* 目前线程块内的warp编号
  int lane = tid % WARP_SIZE; //* 目前线程块内的warp内的线程编号
  // perform warp sync reduce.
  prod = warp_reduce_sum_f32<WARP_SIZE>(prod);
  // warp leaders store the data to shared memory.
  if (lane == 0)
    reduce_smem[warp] = prod;
  __syncthreads(); // make sure the data is in shared memory.
  // the first warp compute the final sum.
  prod = (lane < NUM_WARPS) ? reduce_smem[lane] : 0.0f;
  if (warp == 0)
    prod = warp_reduce_sum_f32<NUM_WARPS>(prod);
  if (tid == 0)
    atomicAdd(y, prod);
}

// Dot Product + Vec4
// grid(N/256), block(256/4)
// a: Nx1, b: Nx1, y=sum(elementwise_mul(a,b))
template <const int NUM_THREADS = 256 / 4>
__global__ void dot_prod_f32x4_f32_kernel(float *a, float *b, float *y, int N) {
  int tid = threadIdx.x;
  int idx = (blockIdx.x * NUM_THREADS + tid) * 4;
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  __shared__ float reduce_smem[NUM_WARPS];

  float4 reg_a = FLOAT4(a[idx]);
  float4 reg_b = FLOAT4(b[idx]);
  float prod = (idx < N) ? (reg_a.x * reg_b.x + reg_a.y * reg_b.y +
                            reg_a.z * reg_b.z + reg_a.w * reg_b.w)
                         : 0.0f;
  int warp = tid / WARP_SIZE;
  int lane = tid % WARP_SIZE;
  // perform warp sync reduce.
  prod = warp_reduce_sum_f32<WARP_SIZE>(prod);
  // warp leaders store the data to shared memory.
  if (lane == 0)
    reduce_smem[warp] = prod;
  __syncthreads(); // make sure the data is in shared memory.
  // the first warp compute the final sum.
  prod = (lane < NUM_WARPS) ? reduce_smem[lane] : 0.0f;
  if (warp == 0)
    prod = warp_reduce_sum_f32<NUM_WARPS>(prod);
  if (tid == 0)
    atomicAdd(y, prod);
}

// -------------------------------------- FP16
// -------------------------------------- Warp Reduce Sum: Half
template <const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ half warp_reduce_sum_f16_f16(half val) {
#pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    //* __hadd() 在半精度数上执行加法操作, 效率比直接使用加法高
    val = __hadd(val, __shfl_xor_sync(0xffffffff, val, mask));
    // val += __shfl_xor_sync(0xffffffff, val, mask);
  }
  return val;
}

template <const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f16_f32(half val) {
  float val_f32 = __half2float(val);
#pragma unroll
  for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
    val_f32 += __shfl_xor_sync(0xffffffff, val_f32, mask);
  }
  return val_f32;
}

template <const int NUM_THREADS = 256>
__global__ void dot_prod_f16_f32_kernel(half *a, half *b, float *y, int N) {
  int tid = threadIdx.x;
  int idx = blockIdx.x * NUM_THREADS + tid;
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  __shared__ float reduce_smem[NUM_WARPS];

  // keep the data in register is enough for warp operaion.
  half prod_f16 = (idx < N) ? __hmul(a[idx], b[idx]) : __float2half(0.0f);
  int warp = tid / WARP_SIZE;
  int lane = tid % WARP_SIZE;
  // perform warp sync reduce.
  float prod = warp_reduce_sum_f16_f32<WARP_SIZE>(prod_f16);
  // warp leaders store the data to shared memory.
  if (lane == 0)
    reduce_smem[warp] = prod;
  __syncthreads(); // make sure the data is in shared memory.
  // the first warp compute the final sum.
  prod = (lane < NUM_WARPS) ? reduce_smem[lane] : 0.0f;
  if (warp == 0)
    prod = warp_reduce_sum_f32<NUM_WARPS>(prod);
  if (tid == 0)
    atomicAdd(y, prod);
}

template <const int NUM_THREADS = 256 / 2>
__global__ void dot_prod_f16x2_f32_kernel(half *a, half *b, float *y, int N) {
  int tid = threadIdx.x;
  int idx = (blockIdx.x * NUM_THREADS + tid) * 2; // 2 half elements per thread
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  __shared__ float reduce_smem[NUM_WARPS];

  // keep the data in register is enough for warp operaion.
  half2 reg_a = HALF2(a[idx]);
  half2 reg_b = HALF2(b[idx]);
  half prod_f16 =
      (idx < N) ? __hadd(__hmul(reg_a.x, reg_b.x), __hmul(reg_a.y, reg_b.y))
                : __float2half(0.0f);
  int warp = tid / WARP_SIZE;
  int lane = tid % WARP_SIZE;
  // perform warp sync reduce.
  float prod = warp_reduce_sum_f16_f32<WARP_SIZE>(prod_f16);
  // warp leaders store the data to shared memory.
  if (lane == 0)
    reduce_smem[warp] = prod;
  __syncthreads(); // make sure the data is in shared memory.
  // the first warp compute the final sum.
  prod = (lane < NUM_WARPS) ? reduce_smem[lane] : 0.0f;
  if (warp == 0)
    prod = warp_reduce_sum_f32<NUM_WARPS>(prod);
  if (tid == 0)
    atomicAdd(y, prod);
}

template <const int NUM_THREADS = 256 / 8>
__global__ void dot_prod_f16x8_pack_f32_kernel(half *a, half *b, float *y,
                                               int N) {
  int tid = threadIdx.x;
  int idx = (blockIdx.x * NUM_THREADS + tid) * 8; // 8 half elements per thread
  constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
  __shared__ float reduce_smem[NUM_WARPS];
  // temporary register(memory), .local space in ptx, addressable
  half pack_a[8], pack_b[8];                    // 8x16 bits=128 bits.
  LDST128BITS(pack_a[0]) = LDST128BITS(a[idx]); // load 128 bits
  LDST128BITS(pack_b[0]) = LDST128BITS(b[idx]); // load 128 bits
  const half z = __float2half(0.0f);

  half prod_f16 = z;
#pragma unroll
  for (int i = 0; i < 8; i += 2) {
    half2 v = __hmul2(HALF2(pack_a[i]), HALF2(pack_b[i]));
    prod_f16 += (((idx + i) < N) ? (v.x + v.y) : z);
  }

  int warp = tid / WARP_SIZE;
  int lane = tid % WARP_SIZE;
  // perform warp sync reduce.
  float prod = warp_reduce_sum_f16_f32<WARP_SIZE>(prod_f16);
  // warp leaders store the data to shared memory.
  if (lane == 0)
    reduce_smem[warp] = prod;
  __syncthreads(); // make sure the data is in shared memory.
  // the first warp compute the final sum.
  prod = (lane < NUM_WARPS) ? reduce_smem[lane] : 0.0f;
  if (warp == 0)
    prod = warp_reduce_sum_f32<NUM_WARPS>(prod);
  if (tid == 0)
    atomicAdd(y, prod);
}

// --------------------- PyTorch bindings for custom kernel
// -----------------------
#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)                                   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                                   \
  if (((T).options().dtype() != (th_type))) {                                  \
    std::cout << "Tensor Info:" << (T).options() << std::endl;                 \
    throw std::runtime_error("values must be " #th_type);                      \
  }

#define LANUCH_DOT_PROD_KERNEL(NT, packed_type, acc_type, element_type)        \
  dot_prod_##packed_type##_##acc_type##_kernel<(NT)>                           \
      <<<grid, block>>>(reinterpret_cast<element_type *>(a.data_ptr()),        \
                        reinterpret_cast<element_type *>(b.data_ptr()),        \
                        prod.data_ptr<float>(), N);

#define DISPATCH_DOT_PROD_KERNEL(K, packed_type, acc_type, element_type,       \
                                 n_elements)                                   \
  const int NT = (K) / (n_elements);                                           \
  dim3 block(NT);                                                              \
  dim3 grid((S));                                                              \
  switch (NT) {                                                                \
  case 32:                                                                     \
    LANUCH_DOT_PROD_KERNEL(32, packed_type, acc_type, element_type)            \
    break;                                                                     \
  case 64:                                                                     \
    LANUCH_DOT_PROD_KERNEL(64, packed_type, acc_type, element_type)            \
    break;                                                                     \
  case 128:                                                                    \
    LANUCH_DOT_PROD_KERNEL(128, packed_type, acc_type, element_type)           \
    break;                                                                     \
  case 256:                                                                    \
    LANUCH_DOT_PROD_KERNEL(256, packed_type, acc_type, element_type)           \
    break;                                                                     \
  case 512:                                                                    \
    LANUCH_DOT_PROD_KERNEL(512, packed_type, acc_type, element_type)           \
    break;                                                                     \
  case 1024:                                                                   \
    LANUCH_DOT_PROD_KERNEL(1024, packed_type, acc_type, element_type)          \
    break;                                                                     \
  default:                                                                     \
    throw std::runtime_error(                                                  \
        "only support (K)/(n_elements): 32/64/128/256/512/1024");              \
    break;                                                                     \
  }

#define TORCH_BINDING_DOT_PROD(packed_type, acc_type, th_type, element_type,   \
                               n_elements)                                     \
  torch::Tensor dot_prod_##packed_type##_##acc_type(torch::Tensor a,           \
                                                    torch::Tensor b) {         \
    CHECK_TORCH_TENSOR_DTYPE(a, (th_type))                                     \
    CHECK_TORCH_TENSOR_DTYPE(b, (th_type))                                     \
    auto options =                                                             \
        torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, 0); \
    auto prod = torch::zeros({1}, options);                                    \
    const int ndim = a.dim();                                                  \
    if (ndim != 2) {                                                           \
      int N = 1;                                                               \
      for (int i = 0; i < ndim; ++i) {                                         \
        N *= a.size(i);                                                        \
      }                                                                        \
      dim3 block(256);                                                         \
      dim3 grid(((N + 256 - 1) / 256) / (n_elements));                         \
      dot_prod_##packed_type##_##acc_type##_kernel<256>                        \
          <<<grid, block>>>(reinterpret_cast<element_type *>(a.data_ptr()),    \
                            reinterpret_cast<element_type *>(b.data_ptr()),    \
                            prod.data_ptr<float>(), N);                        \
    } else {                                                                   \
      const int S = a.size(0);                                                 \
      const int K = a.size(1);                                                 \
      const int N = S * K;                                                     \
      if ((K / (n_elements)) <= 1024) {                                        \
        DISPATCH_DOT_PROD_KERNEL(K, packed_type, acc_type, element_type,       \
                                 n_elements)                                   \
      } else {                                                                 \
        int N = 1;                                                             \
        for (int i = 0; i < ndim; ++i) {                                       \
          N *= a.size(i);                                                      \
        }                                                                      \
        dim3 block(256);                                                       \
        dim3 grid(((N + 256 - 1) / 256) / (n_elements));                       \
        dot_prod_##packed_type##_##acc_type##_kernel<256>                      \
            <<<grid, block>>>(reinterpret_cast<element_type *>(a.data_ptr()),  \
                              reinterpret_cast<element_type *>(b.data_ptr()),  \
                              prod.data_ptr<float>(), N);                      \
      }                                                                        \
    }                                                                          \
    return prod;                                                               \
  }

// packed_type, acc_type, th_type, element_type, n_elements_per_pack
TORCH_BINDING_DOT_PROD(f32, f32, torch::kFloat32, float, 1)
TORCH_BINDING_DOT_PROD(f32x4, f32, torch::kFloat32, float, 4)
TORCH_BINDING_DOT_PROD(f16, f32, torch::kHalf, half, 1)
TORCH_BINDING_DOT_PROD(f16x2, f32, torch::kHalf, half, 2)
TORCH_BINDING_DOT_PROD(f16x8_pack, f32, torch::kHalf, half, 8)

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(dot_prod_f32_f32)
  TORCH_BINDING_COMMON_EXTENSION(dot_prod_f32x4_f32)
  TORCH_BINDING_COMMON_EXTENSION(dot_prod_f16_f32)
  TORCH_BINDING_COMMON_EXTENSION(dot_prod_f16x2_f32)
  TORCH_BINDING_COMMON_EXTENSION(dot_prod_f16x8_pack_f32)
}
